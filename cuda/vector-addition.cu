#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

#define N (256*256*256)
#define BLOCK_SIZE 256

__global__
void vecadd(float *x, float *y, float *z, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        z[i] = x[i] + y[i];
    }
}

int main() {
    int n = N; // N = 256*256*256
    float *h_x = (float*) malloc(n * sizeof(float));
    float *h_y = (float*) malloc(n * sizeof(float));
    float *h_z = (float*) malloc(n * sizeof(float));

    for (int i = 0; i < n; i++) {
        h_x[i] = drand48();
        h_y[i] = 1.0f - h_x[i];  // So that x + y = 1.0
    }

    float *d_x, *d_y, *d_z;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_z, n * sizeof(float));

    hipEvent_t start, stop;
    float time_with_copy = 0.0f, time_kernel_only = 0.0f;

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);



    // ----------- Start timer for copy + kernel ---------------
    hipEventRecord(start, 0);

    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    vecadd<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, d_z, n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    const char *msg = hipGetErrorName(err);
    printf("error = |%s|\n", msg);

    hipMemcpy(h_z, d_z, n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    // ------------ Stop timer for copy + kernel ----------------------





    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_with_copy, start, stop);

    // ------------- Memory Copy -------------------------------
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);




    // ------------- Start timer for kernel only ----------------------------
    hipEventRecord(start, 0);

    vecadd<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, d_z, n);
    hipDeviceSynchronize();
    printf("error = |%s|\n", msg);
    hipEventRecord(stop, 0);
    // ------------- Stop timer for kernel -------------------------------




    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_kernel_only, start, stop);

    // Copy back again to make sure kernel ran
    hipMemcpy(h_z, d_z, n * sizeof(float), hipMemcpyDeviceToHost);

    // Compute max absolute error
    float maxerr = 0.0f;
    for (int i = 0; i < n; i++) {
        maxerr = fmaxf(maxerr, fabs(1.0f - h_z[i]));
    }

    std::cout << "Max absolute error: " << maxerr << std::endl;
    std::cout << "Time (kernel only): " << time_kernel_only << " ms" << std::endl;
    std::cout << "Time (with copy): " << time_with_copy << " ms" << std::endl;

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
